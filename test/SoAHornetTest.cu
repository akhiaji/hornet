#include <iostream>
#include <Core/SoADataLayout/Hornet.cuh>

int main(int argc, char *argv[]) {
    hipSetDevice(1);
    using namespace hornet;  //0  1  2
    std::vector<int> offset = {0, 5, 7, 10};
    std::vector<int> edges = {0, 1, 2, 3, 4, 5, 6, 7, 8, 9};
    int nV = offset.size() - 1;
    int nE = edges.size();
    HornetInit<EMPTY, TypeList<float>, int, int> init(nV, nE, offset.data(), edges.data());
    gpu::Hornet<EMPTY, TypeList<float>, int, int> g(init);
    g.print();
                            //  
    std::vector<int>   h_src = {0, 0, 1, 1, 2, 0, 2, 2, 2, 1};
    std::vector<int>   h_dst = {2, 2, 0, 3, 1, 1, 4, 2, 3, 0};
    std::vector<float> h_wgt = {0, 1, 2, 3, 4, 5, 6, 7, 8, 9};
    thrust::device_vector<int> src   = h_src;
    thrust::device_vector<int> dst   = h_dst;
    thrust::device_vector<float> wgt = h_wgt;
    BatchUpdatePtr<TypeList<int, int, float>> ptr(src.size(), src.data().get(), dst.data().get(), wgt.data().get());
    gpu::BatchUpdate<TypeList<int, int, float>, int> batch(ptr);

    g.insert(batch);
    g.print();
//    std::cerr<<"return 0\n";
    return 0;
}
